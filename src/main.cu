#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include <fstream>
#include <string>
#include <chrono>



using namespace cv;

__global__ 
void vegetationKernel(float *in, float *out, int row, int col, int channel);

__global__ 
void detectionKernel(float *in, int8_t *out, int row, int col);

// Get pixel offset in a 2D matrix at position (i, j)
// row-major memory layout
__host__ __device__
int offset2D(int i, int j, int col) {
    return i * col + j;
}

// Get pixel offset in a 3D matrix at position (i, j, k)
// row-major memory layout, but channel is the innermost loop increment unit 
__host__ __device__
int offset3D(int i, int j, int k, int col, int channel) {
    return (i * col + j) * channel + k; 
}


/************************************************************************************************************** 
 *                                                 C++ version 
 *************************************************************************************************************/

// Calculate vegetation index
void getVegetationIndex(float *img, float *out, int row, int col, int channel) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            float b2 = img[offset3D(i, j, 1, col, channel)];
            float b3 = img[offset3D(i, j, 2, col, channel)];

            if (b3 + b2 == 0) {
                out[offset2D(i, j, col)] = 0;
            } 
            else {
                out[offset2D(i, j, col)] = (b3 - b2) / (b3 + b2);
            }
        }
    }

    return;
}


// Calculate vegetation detection 
void getVegetationDetection(float *veg, int8_t *out, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            out[offset2D(i, j, col)] = veg[offset2D(i, j, col)] > 0.1 ? (int8_t)1 : (int8_t)0;
        }
    }

    return;
} 


/************************************************************************************************************** 
 *                                                 CUDA version 
 *************************************************************************************************************/

void getVegetationIndexCUDA(float *h_img, float *h_veg, int row, int col, int channel) {
    int img_size = row * col * channel * sizeof(float);
    int out_size = row * col * sizeof(float);

    float *d_img, *d_out;

    hipError_t err1 =  hipMalloc((void **) &d_img, img_size);
    hipError_t err2 =  hipMalloc((void **) &d_out, out_size);

    if (err1 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    if (err2 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_img, h_img, img_size, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(ceil(row/32.0), ceil(col/32.0), 1);   
    dim3 threadsPerBlock(32, 32, 1); // 1024 threads per block

    vegetationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_img, d_out, row, col, channel);

    hipMemcpy(h_veg, d_out, out_size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();

    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }

    hipFree(d_img); 
    hipFree(d_out);
}


void getVegetationDetectionCUDA(float *h_veg, int8_t *h_out, int row, int col) {
    int veg_size = row * col * sizeof(float);
    int out_size = row * col * sizeof(int8_t);
    float *d_veg;
    int8_t *d_out;

    hipError_t err1 =  hipMalloc((void **) &d_veg, veg_size);
    hipError_t err2 =  hipMalloc((void **) &d_out, out_size);

    if (err1 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    if (err2 != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_veg, h_veg, veg_size, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(ceil(row/32.0), ceil(col/32.0), 1);   
    dim3 threadsPerBlock(32, 32, 1); // 1024 threads per block

    detectionKernel<<<blocksPerGrid, threadsPerBlock>>>(d_veg, d_out, row, col);

    hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();

    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }

    hipFree(d_veg); 
    hipFree(d_out);
} 


/************************************************************************************************************** 
 *                                                 Driver code 
 *************************************************************************************************************/
void printImg(float *d, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            std::cout << d[offset2D(i, j, col)] << " ";
        }
        std::cout << std::endl;
    }
}


void printImg(int8_t *d, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            std::cout << (int)d[offset2D(i, j, col)] << " ";
        }
        std::cout << std::endl;
    }
}

void printCube(float *d, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            std::cout << "( " << d[offset3D(i, j, 0, col, 3)] << " " << d[offset3D(i, j, 1, col, 3)] << " " << d[offset3D(i, j, 2, col, 3)] << " )"; 
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
}

int main() 
{ 
    // Image metadata is hard coded for now, but can be passed as arguments to main
    int height = 6058;
    int width = 3320; 
    int channel= 3;

    std::cout << "width " << width << " height " << height << std::endl;

    // Read from tmp .txt file
    int img_size = width * height * channel;
    float *dat = new float[img_size]();

    std::ifstream file("../tmp.txt");
    std::string str; 

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            for (int k = 0; k < channel; k++) {
                std::getline(file, str);
                dat[offset3D(i, j, k, width, channel)] = std::atof(str.c_str());
            }
        }
    }

    // // verify correct

    // for (int i = 0; i < 1; i++) {
    //     for (int j = 0; j < 1; j++) {
    //         for (int k = 0; k < channel; k++) {
    //             std::cout << dat[offset3D(i, j, k, width, channel)] << " ";
    //         }
    //     }
    // }


    // C++ version
    int out_size = height * width;

    std::cout << "C++ ======================================\n";
    float *veg_cpp = new float[out_size]();
    int8_t *det_cpp = new int8_t[out_size]();
    
    // time veg in cpp
    auto t1 = std::chrono::high_resolution_clock::now();

    getVegetationIndex(dat, veg_cpp, height, width, channel);

    auto t2 = std::chrono::high_resolution_clock::now();

    auto veg_cpp_duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();

    // time det in cpp
    t1 = std::chrono::high_resolution_clock::now();

    getVegetationDetection(veg_cpp, det_cpp, height, width);

    t2 = std::chrono::high_resolution_clock::now();

    auto det_cpp_duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();

    // verify results in cpp
    std::cout << "Veg time : cpp " << veg_cpp_duration << std::endl;
    std::cout << "Det time : cpp " << det_cpp_duration << std::endl;

    std::cout << "Veg results : cpp " << std::endl;
    printImg(veg_cpp, 5, 5);
    std::cout << "Det results : cpp " << std::endl;
    printImg(det_cpp, 5, 5);

    // CUDA version
    std::cout << "CUDA ======================================\n";

    float *veg_cuda = new float[out_size]();
    int8_t *det_cuda = new int8_t[out_size]();

    // time veg in cuda 
    t1 = std::chrono::high_resolution_clock::now();

    getVegetationIndexCUDA(dat, veg_cuda, height, width, channel);

    t2 = std::chrono::high_resolution_clock::now();

    auto veg_cuda_duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();

    // time det in cuda 
    t1 = std::chrono::high_resolution_clock::now();

    getVegetationDetectionCUDA(veg_cuda, det_cuda, height, width);

    t2 = std::chrono::high_resolution_clock::now();

    auto det_cuda_duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();

    // verify results in cuda 
    std::cout << "Veg time : cuda " << veg_cuda_duration << std::endl;
    std::cout << "Det time : cuda " << det_cuda_duration << std::endl;

    std::cout << "Veg results : cuda" << std::endl;
    printImg(veg_cuda, 5, 5);
    std::cout << "Det results : cuda" << std::endl;
    printImg(det_cuda, 5, 5);

    // save results
    std::cout << "Write results to image ...\n";
    cv::imwrite("../veg_index.bmp",  cv::Mat(height, width, CV_32FC1, veg_cpp));
    cv::imwrite("../veg_detection.tif",  cv::Mat(height, width, CV_8UC1, det_cpp));

    return 0; 
} 



/************************************************************************************************************** 
 *                                                 CUDA kernel 
 *************************************************************************************************************/

__global__ 
void vegetationKernel(float *in, float *out, int row, int col, int channel){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < row && j < col) {
        float a = (in[offset3D(i, j, 2, col, channel)] - in[offset3D(i, j, 1, col, channel)] ) ;
        float b = (in[offset3D(i, j, 2, col, channel)] + in[offset3D(i, j, 1, col, channel)] ) ;

        if (b != 0) {
            out[offset2D(i, j, col)] = a / b;
        }
        else {
            out[offset2D(i, j, col)] = 0;
        } 
    } 
}


__global__ 
void detectionKernel(float *in, int8_t *out, int row, int col){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < row && j < col) {
        out[offset2D(i, j, col)] = in[offset2D(i, j, col)] > 0.1 ? 1 : 0;
    } 
}